#include "LinearSolver.cuh"


__device__ int matrixToArray(int row, int col, int n)
{
	return (row * n + col);
}

__device__ void matrixVectorProduct(complex_t *M, complex_t *v, complex_t *y, int n)
{
	for (int i = 0; i < n; i++)
	{
		*(y + i) = 0;
		for (int j = 0; j < n; j++)
		{
			*(y + i) += *(M + matrixToArray(i, j, n)) * *(v + j);
		}
	}
}

__device__ void matrixMatrixProduct(complex_t *A, complex_t *B, complex_t *C, int n)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			complex_t currentSum = complex_t(0, 0);
			for (int k = 0; k < n; k++)
			{
				currentSum += *(A + matrixToArray(i, k, n)) * *(B + matrixToArray(k, j, n));
			}
			*(C + matrixToArray(i, j, n)) = currentSum;
		}
	}
}


// Decomposes Matrix A into an Upper (stored in U) and a Lower (stored in L) triangular matrix using
// doolittle factorization.
//
// n is the number of rows/columns of A, A is a row-major matrix.
//
// Code implemented from pesudocode found in Cheney/Kincade, "Numerical Mathematics", pg 300.
__device__ void LUDecomp(complex_t *A, complex_t *L, complex_t *U, int n)
{

	//printf("A(device) = \n");
	for (int row = 0; row < n; row++)
	{
		for (int col = 0; col < n; col++)
		{
			//printf("%0.2f ", *(A + matrixToArray(row, col, n)));
			*(L + matrixToArray(row, col, n)) = complex_t(0, 0);
			*(U + matrixToArray(row, col, n)) = complex_t(0, 0);
		}
		//printf("\n");
	}

	for (int k = 0; k < n; k++)
	{
		// L_kk = 1 (set 1 along the diagonal of L)
		*(L + matrixToArray(k, k, n)) = 1;

		for (int j = k; j < n; j++)
		{
			complex_t currentSum = 0;
			for (int s = 0; s < k; s++)
			{
				complex_t L_Val = *(L + matrixToArray(k, s, n));
				complex_t U_Val = *(U + matrixToArray(s, j, n));;
				currentSum += L_Val * U_Val;
			}

			// U_kj = A_kj - currentSum
			*(U + matrixToArray(k, j, n)) = *(A + matrixToArray(k, j, n)) - currentSum;
		}

		for (int i = k + 1; i < n; i++)
		{
			complex_t currentSum = 0;
			for (int s = 0; s < k; s++)
			{
				complex_t L_Val = *(L + matrixToArray(i, s, n));
				complex_t U_Val = *(U + matrixToArray(s, k, n));;
				currentSum += L_Val * U_Val;
			}

			// L_ik = (A_ik - currentSum)/U_kk
			*(L + matrixToArray(i, k, n)) = (*(A + matrixToArray(i, k, n)) - currentSum) / *(U + matrixToArray(k, k, n));
		}

	}
}


__device__ void matrixVectorProduct(double *M, double *v, double *y, int n)
{
	for (int i = 0; i < n; i++)
	{
		*(y + i) = 0;
		for (int j = 0; j < n; j++)
		{
			*(y + i) += *(M + matrixToArray(i, j, n)) * *(v + j);
		}
	}
}

// Solves Ax = b for x, where A has been decomposed into upper (U) and lower (L) triangular matrices.
//
//	LUx = b -> first, let Ux = z => Lz = b, solve for z.
// 
//	Second, solve Ux = z for x.
// Code written from pseudocode in Cheney/Kincaid, pg 301.
__device__ void linSolve(complex_t *A, complex_t *L, complex_t *U, complex_t *b, int n)
{
	complex_t *x = &A[0];
	complex_t *z = &A[n];

	for (int i = 0; i < n; i++)
	{
		*(z + i) = 0;
	}
	// *******************
	// Solve Lz = b for z.
	// *******************

	// z_0 = b_0.
	*(z) = *(b);

	// finding the rest of z 
	for (int i = 1; i < n; i++)
	{
		complex_t currentSum = 0;
		for (int j = 0; j < i; j++)
		{
			complex_t lVal = *(L + matrixToArray(i, j, n));
			complex_t zVal = *(z + j);
			currentSum += lVal * zVal;
		}

		*(z + i) = *(b + i) - currentSum;
	}

	// *******************
	// Solve Ux = z for x.
	// *******************

	//x_n = z_n/U_nn
	*(x + (n - 1)) = *(z + (n - 1)) / *(U + matrixToArray(n - 1, n - 1, n));

	// finding the rest of x
	for (int i = (n - 2); i >= 0; i--)
	{
		complex_t currentSum = 0;
		for (int j = i + 1; j < n; j++)
		{
			complex_t uVal = *(U + matrixToArray(i, j, n));
			complex_t xVal = *(x + j);
			currentSum += uVal * xVal;
		}

		*(x + i) = (*(z + i) - currentSum) / *(U + matrixToArray(i, i, n));
	}

}

__device__ void LUDecomp_PartialPivot(complex_t *A, complex_t *L, complex_t *U, complex_t *P, int n)
{

	//printf("A(device) = \n");
	for (int row = 0; row < n; row++)
	{
		for (int col = 0; col < n; col++)
		{
			//printf("%0.2f ", *(A + matrixToArray(row, col, n)));
			*(L + matrixToArray(row, col, n)) = complex_t(0, 0);
			*(U + matrixToArray(row, col, n)) = complex_t(0, 0);
			*(P + matrixToArray(row, col, n)) = complex_t(0, 0);
		}
	}

	for (int k = 0; k < n; k++)
	{
		*(P + matrixToArray(k, k, n)) = complex_t(1, 0);
	}

	int pivotRow;
	complex_t pivotVal;
	complex_t pivotCandidate;

	complex_t tempP;
	complex_t tempA;
	for (int k = 0; k < n; k++)
	{
		// L_kk = 1 (set 1 along the diagonal of L)
		*(L + matrixToArray(k, k, n)) = complex_t(1, 0);

		// Pivoting
		pivotRow = k;
		pivotVal = *(A + matrixToArray(k, k, n));
		double pivotNorm = pivotVal.real() * pivotVal.real() + pivotVal.imag() * pivotVal.imag();
		for (int i = k + 1; i < n; i++)
		{
			pivotCandidate = *(A + matrixToArray(i, k, n));
			double candidateNorm = pivotCandidate.real() * pivotCandidate.real() + pivotCandidate.imag() * pivotCandidate.imag();
			if (candidateNorm > pivotNorm)
			{
				pivotVal = pivotCandidate;
				pivotRow = i;
				pivotNorm = candidateNorm;
			}
		}

		// Swapping rows if necessary.
		if (pivotRow != k)
		{
			for (int j = 0; j < n; j++)
			{
				// swap rows pivotRow and k of P
				tempP = *(P + matrixToArray(pivotRow, j, n));
				*(P + matrixToArray(pivotRow, j, n)) = *(P + matrixToArray(k, j, n));
				*(P + matrixToArray(k, j, n)) = tempP;

				// swap rows pivotRow and k of A
				tempA = *(A + matrixToArray(pivotRow, j, n));
				*(A + matrixToArray(pivotRow, j, n)) = *(A + matrixToArray(k, j, n));
				*(A + matrixToArray(k, j, n)) = tempA;
			}

			if (k >= 1)
			{
				// interchange rows pivotRow and k in cols 1:k-1 of L
				for (int j = 0; j < k; j++)
				{
					tempA = *(L + matrixToArray(pivotRow, j, n));
					*(L + matrixToArray(pivotRow, j, n)) = *(L + matrixToArray(k, j, n));
					*(L + matrixToArray(k, j, n)) = tempA;
				}

			}
		}

		// Peforming gaussian elimination
		for (int i = k + 1; i < n; i++)
		{
			// L[i][k] = A[i][k] / A[k][k]
			*(L + matrixToArray(i, k, n)) = *(A + matrixToArray(i, k, n)) / *(A + matrixToArray(k, k, n));

			for (int j = k + 1; j < n; j++)
			{
				// A[i][j] = A[i][j] - L[i][k] * A[k][j]
				*(A + matrixToArray(i, j, n)) -= *(L + matrixToArray(i, k, n)) * *(A + matrixToArray(k, j, n));
			}
		}

		for (int j = k; j < n; j++)
		{
			//U[k][j] = A[k][j];
			*(U + matrixToArray(k, j, n)) = *(A + matrixToArray(k, j, n));
		}
	}
}


// Solves the linear system Ax = b.
// Stores the value of x in A from 0:n
__device__ void solveLinearSystem(complex_t *A, complex_t *L, complex_t *U, complex_t *P, complex_t *b, int n, double t)
{

	LUDecomp_PartialPivot(A, L, U, P, n);
	//if (threadIdx.x == 0 && t == 1.0)
	//{
	//	printf("A = \n");
	//	for (int i = 0; i < n; i++)
	//	{
	//		for (int j = 0; j < n; j++)
	//		{
	//			complex_t currentVal = *(A + (i * n) + j);
	//			printf("(%0.5f,  %0.5f) ", currentVal.real(), currentVal.imag());
	//		}
	//		printf("\n");
	//	}
	//	printf("\n\n");

	//	printf("L = \n");
	//	for (int i = 0; i < n; i++)
	//	{
	//		for (int j = 0; j < n; j++)
	//		{
	//			printf("%f ", *(L + (i * n) + j));
	//		}
	//		printf("\n");
	//	}
	//	printf("\n\n");
	//}

	// Permutating b vec by the permutation matrix P
	matrixVectorProduct(P, b, &A[0], n);

	for (int i = 0; i < n; i++)
	{
		*(b + i) = *(A + i);
	}

	linSolve(A, L, U, b, n);
	// A from 0:n holds the value of x



}